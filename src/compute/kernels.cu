#include "hip/hip_runtime.h"
#include <compute/kernels.hpp>

#include <math/Vector.hpp>
#include <math/Matrix.hpp>

using count_t = int;

#define FOR(var, range)\
	for (count_t var = blockDim.x * blockIdx.x + threadIdx.x; \
	     (var) < (range);                                     \
	     (var) += gridDim.x * blockDim.x                      \
 )

__global__ void kHelloWorld(int value)
{
	printf("[%d][%d] Hello world x%d!\n", blockIdx.x, threadIdx.x, value);
}

__global__ void kPos2DToTransform3x3(count_t count, const Vec2f* position, const float* radius, Mat3x3f* outTransform)
{
	FOR(i, count) {
		Mat3x3f transform = {Mat3x3f::IdentityInitT{}, 1.0f};
		transform[0][0] = radius[i];
		transform[1][1] = radius[i];
		transform.refColumn<2>() = Vec3f(position[i], 1.0f);
		outTransform[i] = transform;
	}
}

__global__ void kApplyVelocity(count_t count, float dt, const Vec2f* vel, Vec2f* pos)
{
	FOR(i, count) {
		pos[i] += dt * vel[i];
	}
}

__global__ void kGameOfLife(unsigned char* in, unsigned char* out, int width, int height)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int x = tid % width;
	int y = tid / width;
	if (x >= width || y >= height) {
		return;
	}

	unsigned char currValue;
	unsigned char count = 0;
	for (int dy = -1; dy <= 1; ++dy) {
		for (int dx = -1; dx <= 1; ++dx) {
			if (!(0 <= x + dx && x + dx < width)) {
				continue;
			}
			if (!(0 <= y + dy && y + dy < height)) {
				continue;
			}
			if (dx == 0 && dy == 0) {
				currValue = in[y*width + x];
				continue;
			}
			count += (in[width * (y+dy) + (x+dx)] == 255) ? 1 : 0;
		}
	}
	unsigned char nextValue = currValue;
	if (currValue < 255 && count == 3) {
		nextValue = 255; // becomes alive
	}
	if (currValue == 255) {
		bool stayAlive = (count == 2 || count == 3);
		nextValue = stayAlive ? 255 : 100;
	}
	if (nextValue < 255 && nextValue > 0) {
		nextValue -= 2;
	}

	out[y * width + x] = nextValue;
}

// TODO: tmp workaround, remove me
__global__ void kSplit(unsigned char* data)
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned char* v = &data[tid];
	*v = *v > 127 ? 255 : 0;
}

// __device__ float dNormalizeU8(uint8_t in)
// {
// 	return (static_cast<float>(in) / 255.0f);
// }

// __device__ float dIdentity(float in)
// {
// 	return in;
// }

// __device__ float (*dPtrNormalizeU8)(uint8_t) = dNormalizeU8;
//
// template<typename InputType>
// __global__ void kColorizeBlackToWhite(count_t count, InputType* in, color_t* out, float (*normalize)(InputType))
// {
// 	int tid = threadIdx.x + blockIdx.x * blockDim.x;
// 	if (tid >= count) {
// 		return;
// 	}
// 	// TODO: test it (does it cover whole range?)
// 	uint8_t value = static_cast<uint8_t>(255.0f * normalize(in[tid]));
// 	out[tid].r = value;
// 	out[tid].g = value;
// 	out[tid].b = value;
// 	out[tid].a = 255;
// }
//
// template __global__ void kColorizeBlackToWhite<uint8_t>(count_t count, uint8_t* in, color_t* out, float (*normalize)(uint8_t));
// template __global__ void kColorizeBlackToWhite<float>(count_t count, float* in, color_t* out, float (*normalize)(float));

__global__ void kTmpColorizeCustomU8(count_t count, const uint8_t* in, color_t* out)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= count) {
		return;
	}
	bool even = (tid % 2) == 0;
	bool odd = !even;
	if (in[tid] < 128) {
		out[tid].r = (1+even) * in[tid];
		out[tid].b = (1+odd) * in[tid];
		out[tid].g = 0;
	}
	else {
		out[tid].r = 0;
		out[tid].g = in[tid];
		out[tid].b = in[tid];
	}
	out[tid].a = 255;
}

// __global__ void kTmpColorizeCustomF32(count_t count, const float* in, const float* inMin, const float* inMax,  color_t* out)
// {
// 	int tid = threadIdx.x + blockIdx.x * blockDim.x;
// 	if (tid >= count) {
// 		return;
// 	}
// 	// float min = *inMin;
// 	// float max = *inMax;
// 	float min = 0.0f;
// 	float max = 1.0f;
// 	uint8_t value = static_cast<uint8_t>(255 * (in[tid] - min) / (max - min));
// 	out[tid].r = value;
// 	out[tid].g = 0;
// 	out[tid].b = 0;
// 	out[tid].a = 255;
// }

__global__ void kTmpColorizeCustomF32(count_t count, const float* in, const float* inMin, const float* inMax,  color_t* out)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= count) {
		return;
	}
	// float min = *inMin;
	// float max = *inMax;
	float min = 0.0f;
	float max = 1.0f;
	float fval = (in[tid] - min) / (max - min);

	if (fval < 0.2) {
		float progress = (fval - 0.0f) / 0.2f;
		uint8_t uval = static_cast<uint8_t>(255.0f * progress);
		out[tid].r = uval;
		out[tid].g = 0;
		out[tid].b = 0;
	}
	else if (fval < 0.4) {
		float progress = (fval - 0.2f) / 0.2f;
		uint8_t uval = static_cast<uint8_t>(255.0f * progress);
		out[tid].r = 255;
		out[tid].g = uval;
		out[tid].b = 0;
	}
	else if (fval < 0.6) {
		float progress = (fval - 0.4f) / 0.2f;
		uint8_t uval = static_cast<uint8_t>(255.0f * progress);
		out[tid].r = 255-uval;
		out[tid].g = 255;
		out[tid].b = 0;
	}
	else if (fval < 0.8) {
		float progress = (fval - 0.6f) / 0.2f;
		uint8_t uval = static_cast<uint8_t>(255.0f * progress);
		out[tid].r = 0;
		out[tid].g = 255;
		out[tid].b = uval;
	}
	else {
		float progress = (fval - 0.8f) / 0.2f;
		uint8_t uval = static_cast<uint8_t>(255.0f * progress);
		out[tid].r = 0;
		out[tid].g = 255-uval;
		out[tid].b = 255;
	}
	out[tid].a = 255;
}

__global__ void kTmpSetNCube(count_t width, count_t height, float* data, NCube2i rect, float value)
{
	int tX = threadIdx.x + blockIdx.x * blockDim.x;
	int tY = threadIdx.y + blockIdx.y * blockDim.y;
	bool inRange = (tX < width) && (tY < height);
	bool inNCube = (rect.min().x() <= tX) && (tX < rect.max().x())
	            && (rect.min().y() <= tY) && (tY < rect.max().y());
	if (!inRange || !inNCube) {
		return;
	}
	data[tX + tY * width] = value;
}

__global__ void kHeatTransfer(count_t width, count_t height, const float* curr, float* next, float coeff)
{
	int tX = threadIdx.x + blockIdx.x * blockDim.x;
	int tY = threadIdx.y + blockIdx.y * blockDim.y;
	bool inRange = (tX < width) && (tY < height);
	if (!inRange) {
		return;
	}
	bool isBorder = (tX == 0) || (tY == 0) || (tX == width - 1) || (tY == height - 1);
	if (isBorder) {
		next[tX + tY * width] = 0.0f;
		return;
	}

	next[tX + tY * width] = curr[tX + tY * width];
	next[tX + tY * width] += coeff * (curr[(tX+1) + (tY) * width] - curr[tX + tY * width]);
	next[tX + tY * width] += coeff * (curr[(tX-1) + (tY) * width] - curr[tX + tY * width]);
	next[tX + tY * width] += coeff * (curr[(tX) + (tY+1) * width] - curr[tX + tY * width]);
	next[tX + tY * width] += coeff * (curr[(tX) + (tY-1) * width] - curr[tX + tY * width]);
}